#include "hip/hip_runtime.h"
#include "SimpleWertheim.cuh"

#include <stdio.h>

__device__ float _CUDA_X(float rho, float B2, float two_valence_delta) {
	float sqrt_argument = 2.f * two_valence_delta * rho;
	return (sqrt_argument < 1e-3f) ? 1.f - sqrt_argument / 2.f : (-1.f + sqrtf(1.f + 2.f * two_valence_delta * rho)) / (two_valence_delta * rho);
}

__global__ void _compute_der_bulk_free_energy(field_type *rho, float *rho_der, int grid_size, float B2, int valence, float two_valence_delta) {
	if(IND >= grid_size) return;

    float rho_ind = rho[IND];
	float der_f_ref = logf(rho_ind) + 2.f * B2 * rho_ind;
	float X = _CUDA_X(rho_ind, B2, two_valence_delta);
	float der_f_bond = valence * logf(X);

	rho_der[IND] = der_f_ref + der_f_bond;
}

namespace ch {

    void simple_wertheim_der_bulk_free_energy(field_type *rho, float *rho_der, int grid_size, float B2, int valence, float two_valence_delta) {
        const int blocks = grid_size / BLOCK_SIZE + 1;
        _compute_der_bulk_free_energy<<<blocks, BLOCK_SIZE>>>(rho, rho_der, grid_size, B2, valence, two_valence_delta);
    }

}
